#include "hip/hip_runtime.h"
/* Copyright (C) 2010 Ion Torrent Systems, Inc. All Rights Reserved */

// patch for CUDA5.0/GCC4.7
#undef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_USE_INT128

#include <iostream>

#include "JobWrapper.h"
#include "SignalProcessingFitterQueue.h"
#include "GpuMultiFlowFitControl.h"
#include "DarkHalo.h"
using namespace std;


////////////////////////////////////////
// workset (represents one job


WorkSet::WorkSet()
{
  GpuMultiFlowFitControl * fitcontrol = GpuMultiFlowFitControl::Instance();
   
  _fd[0] = fitcontrol->GetMatrixConfig("FitWellAmplBuffering");
  _fd[1] = fitcontrol->GetMatrixConfig("FitWellPostKey");

  _maxFrames = 0; // only set if we don't want to determine the mem sizes for a specific number of frames or no item is set
  _maxBeads = 0; // only set if we don't want to determine the mem sizes for a specific number of beads or no item is set

 _info = NULL;
}

WorkSet::WorkSet(BkgModelWorkInfo * i)
{
   GpuMultiFlowFitControl * fitcontrol = GpuMultiFlowFitControl::Instance();
   
  _fd[0] = fitcontrol->GetMatrixConfig("FitWellAmplBuffering");
  _fd[1] = fitcontrol->GetMatrixConfig("FitWellPostKey");

  _maxFrames = 0; // only set if we don't want to determine the mem sizes for a specific number of frames or no item is set
  _maxBeads = 0; // only set if we don't want to determine the mem sizes for a specific number of beads or no item is set

 _info = i;
}


WorkSet::~WorkSet()
{
}



void WorkSet::setMaxFrames(int frames)
{
  _maxFrames = frames;
}

int WorkSet::getMaxFrames()
{
  return (_maxFrames != 0)?(_maxFrames):(GpuMultiFlowFitControl::GetMaxFrames());
}
void WorkSet::setMaxBeads(int beads)
{
  _maxBeads = beads;
}

int WorkSet::getMaxBeads()
{
  return (_maxBeads != 0)?(_maxBeads):(GpuMultiFlowFitControl::GetMaxBeads());
}

void WorkSet::setData(BkgModelWorkInfo * i)
{
 _info = i;
}

bool WorkSet::isSet()
{
 return (_info != NULL)?(true):(false);
}


int WorkSet::getNumBeads() {  

    if(isSet()){            
      return _info->bkgObj->region_data->my_beads.numLBeads; 
    }
  return getMaxBeads();// GpuMultiFlowFitControl::GetMaxBeads();
}


int WorkSet::getNumFrames() { 

  if(isSet()){
    return _info->bkgObj->region_data->time_c.GetTimeCompressedFrames(); 
  }

  return getMaxFrames();  

}





int WorkSet::getMaxSteps()
{
  GpuMultiFlowFitControl * fitcontrol = GpuMultiFlowFitControl::Instance();
  return fitcontrol->GetMaxSteps();
}

int WorkSet::getMaxParams()
{
  GpuMultiFlowFitControl * fitcontrol = GpuMultiFlowFitControl::Instance();
  return fitcontrol->GetMaxParamsToFit(); 
}

int WorkSet::getNumSteps(int fit_index)
{
  return _fd[fit_index]->GetNumSteps();
}
int WorkSet::getNumParams(int fit_index)
{
  return _fd[fit_index]->GetNumParamsToFit();
}




int WorkSet::getAbsoluteFlowNum() { return _info->bkgObj->region_data->my_flow.buff_flow[0]; }

reg_params * WorkSet::getRegionParams() {   return  &_info->bkgObj->region_data->my_regions.rp;  }
BeadTracker * WorkSet::getBeadTracker(){   return &_info->bkgObj->region_data->my_beads; }
bead_params * WorkSet::getBeadParams(){   return &_info->bkgObj->region_data->my_beads.params_nn[0]; }
bead_state * WorkSet::getBeadState(){   return &_info->bkgObj->region_data->my_beads.all_status[0]; }
float * WorkSet::getEmphVec(){   return &_info->bkgObj->region_data->emphasis_data.emphasis_vector_storage[0]; } 
float * WorkSet::getDarkMatter(){   return &_info->bkgObj->region_data->my_regions.missing_mass.dark_matter_compensator[0]; }
int * WorkSet::getFlowIdxMap(){   return _info->bkgObj->region_data->my_flow.flow_ndx_map; }
FG_BUFFER_TYPE * WorkSet::getFgBuffer(){   return _info->bkgObj->region_data->my_trace.fg_buffers; }
float * WorkSet::getDeltaFrames(){   return &_info->bkgObj->region_data->time_c.deltaFrame[0]; }  
int * WorkSet::getStartNuc(){   return _info->bkgObj->region_data->my_regions.cache_step.i_start_fine_step; }


float * WorkSet::getShiftedBackground(){   
  _info->bkgObj->region_data->my_scratch.FillShiftedBkg (*_info->bkgObj->region_data->emptytrace, _info->bkgObj->region_data->my_regions.rp.tshift, _info->bkgObj->region_data->time_c, true);
  return _info->bkgObj->region_data->my_scratch.shifted_bkg; 
}

float * WorkSet::getCalculateNucRise(){   
  _info->bkgObj->region_data->my_regions.cache_step.CalculateNucRiseFineStep (&_info->bkgObj->region_data->my_regions.rp, _info->bkgObj->region_data->time_c, _info->bkgObj->region_data->my_flow); // the same for the whole region because time-shift happens per well
  return _info->bkgObj->region_data->my_regions.cache_step.nuc_rise_fine_step; 
}

float * WorkSet::getCalculateNucRiseCoarse()
{
  _info->bkgObj->region_data->my_regions.cache_step.CalculateNucRiseCoarseStep (&_info->bkgObj->region_data->my_regions.rp, _info->bkgObj->region_data->time_c, _info->bkgObj->region_data->my_flow);
  return &_info->bkgObj->region_data->my_regions.cache_step.nuc_rise_coarse_step[0];
}

 
void WorkSet::setUpFineEmphasisVectors() {
  _info->bkgObj->region_data->SetFineEmphasisVectors();
}

float WorkSet::getAmpLowLimit() 
{ 
  return _info->bkgObj->getGlobalDefaultsForBkgModel().signal_process_control.AmplLowerLimit;
}

float WorkSet::getkmultLowLimit()
{
  return _info->bkgObj->getGlobalDefaultsForBkgModel().signal_process_control.kmult_low_limit;
}

float WorkSet::getkmultHighLimit()
{
  return _info->bkgObj->getGlobalDefaultsForBkgModel().signal_process_control.kmult_hi_limit;
}

float* WorkSet::getClonalCallScale() 
{
  return _info->bkgObj->getGlobalDefaultsForBkgModel().fitter_defaults.clonal_call_scale;
}

float WorkSet::getClonalCallPenalty() 
{
  return _info->bkgObj->getGlobalDefaultsForBkgModel().fitter_defaults.clonal_call_penalty;
}

int * WorkSet::getStartNucCoarse()
{
  return _info->bkgObj->region_data->my_regions.cache_step.i_start_coarse_step ;
}

bool WorkSet::performAlternatingFit()
{
  return _info->bkgObj->getGlobalDefaultsForBkgModel().signal_process_control.fit_alternate;
}

bound_params * WorkSet::getBeadParamsMax()
{
  return &_info->bkgObj->region_data->my_beads.params_high;
}

 bound_params * WorkSet::getBeadParamsMin()
{
  return &_info->bkgObj->region_data->my_beads.params_low;
}

float WorkSet::getMaxEmphasis()
{
  return _info->bkgObj->region_data->my_beads.max_emphasis;
};

bool WorkSet::useDynamicEmphasis()
{
  return (ChipIdDecoder::GetGlobalChipId() == ChipId900);
}


CpuStep_t* WorkSet::getPartialDerivSteps(int fit_index)
{
  return _fd[fit_index]->GetPartialDerivSteps();
}

unsigned int* WorkSet::getJTJMatrixMap(int fit_index)
{
  return _fd[fit_index]->GetJTJMatrixMapForDotProductComputation();
}

unsigned int* WorkSet::getBeadParamIdxMap(int fit_index)
{
  return _fd[fit_index]->GetParamIdxMap();
}


float * WorkSet::getFrameNumber() { 
  return &_info->bkgObj->region_data->time_c.frameNumber[0]; 
}  


//////////////////////////////////////////////////////////////////////////////////////
///SIZES:

//// N

int WorkSet::getBeadParamsSize(bool padded)
{
	int size = sizeof(bead_params);
  return size*( (!padded)?(getNumBeads()):(getPaddedN()) ); 
}

int WorkSet::getBeadStateSize(bool padded)
{
	int size = sizeof(bead_state);
  return size*( (!padded)?(getNumBeads()):(getPaddedN()) ); 
}

int WorkSet::getFgBufferSize(bool padded)
{
  return getFlxFxB(padded);  
}


int WorkSet::getFgBufferSizeShort(bool padded)
{
	int size = sizeof(FG_BUFFER_TYPE)*getNumFrames()*NUMFB;
  return size*( (!padded)?(getNumBeads()):(getPaddedN()) ); 
}

int WorkSet::getFlxFxB(bool padded)
{
  int size = sizeof(float)*NUMFB*getNumFrames();
  return size*( (!padded)?(getNumBeads()):(getPaddedN()) ); 
}

int WorkSet::getFxB(bool padded)
{
  int size = sizeof(float)*getNumFrames();
  return size*( (!padded)?(getNumBeads()):(getPaddedN()) ); 
}

int WorkSet::getFlxB(bool padded)
{
  int size = sizeof(float)*NUMFB;
  return size*( (!padded)?(getNumBeads()):(getPaddedN()) ); 
}

int WorkSet::getFloatPerBead(bool padded)
{
  int size = sizeof(float);  
  return size*( (!padded)?(getNumBeads()):(getPaddedN()) ); 
}
///// non-N

int WorkSet::getRegionParamsSize(bool padded)
{
	int size = sizeof(reg_params);
	return (!padded)?(size):(padTo128Bytes(size));
}

int WorkSet::getEmphVecSize(bool padded)
{
	int size = sizeof(float)*(MAX_POISSON_TABLE_COL)*getNumFrames();
	return (!padded)?(size):(padTo128Bytes(size));
}
 
int WorkSet::getDarkMatterSize(bool padded)
{
	int size = sizeof(float)*NUMNUC*getNumFrames();
	return (!padded)?(size):(padTo128Bytes(size));
}

int WorkSet::getShiftedBackgroundSize(bool padded)
{
	int size = sizeof(float)*NUMFB*getNumFrames();
	return (!padded)?(size):(padTo128Bytes(size));
}

int WorkSet::getFlowIdxMapSize(bool padded)
{
	int size = sizeof(int)*NUMFB;
	return (!padded)?(size):(padTo128Bytes(size));
}



int WorkSet::getDeltaFramesSize(bool padded)
{
	int size = sizeof(float)*getNumFrames();
	return (!padded)?(size):(padTo128Bytes(size));
}
 
int WorkSet::getNucRiseSize(bool padded)
{
	int size = sizeof(float) * ISIG_SUB_STEPS_SINGLE_FLOW * getNumFrames() * NUMFB;
	return (!padded)?(size):(padTo128Bytes(size));
}

int WorkSet::getStartNucSize(bool padded)
{
	int size = sizeof(int)*NUMFB;
	return (!padded)?(size):(padTo128Bytes(size));
}

int WorkSet::getNucRiseCoarseSize(bool padded)
{
	int size = sizeof(float) * ISIG_SUB_STEPS_MULTI_FLOW * getNumFrames() * NUMFB;
	return (!padded)?(size):(padTo128Bytes(size));
}

int WorkSet::getStartNucCoarseSize(bool padded)
{
	int size = sizeof(int)*NUMFB;
	return (!padded)?(size):(padTo128Bytes(size));
}

int WorkSet::getBeadParamsMaxSize(bool padded)
{
	int size = sizeof(bound_params);
	return (!padded)?(size):(padTo128Bytes(size));
}

int WorkSet::getBeadParamsMinSize(bool padded)
{
	int size = sizeof(bound_params);
	return (!padded)?(size):(padTo128Bytes(size));
}

int WorkSet::getClonalCallScaleSize(bool padded)
{
	int size = sizeof(float)*MAGIC_CLONAL_CALL_ARRAY_SIZE;
	return (!padded)?(size):(padTo128Bytes(size));
}
 

int WorkSet::getPartialDerivStepsSize(int fit_index, bool padded)
{
  int size = sizeof(CpuStep_t)*_fd[fit_index]->GetNumSteps();
  return (!padded)?(size):(padTo128Bytes(size));
}

int WorkSet::getJTJMatrixMapSize(int fit_index, bool padded)
{
  int size = sizeof(unsigned int) * _fd[fit_index]->GetNumParamsToFit()*_fd[fit_index]->GetNumParamsToFit();  
  return (!padded)?(size):(padTo128Bytes(size));
}

int WorkSet::getBeadParamIdxMapSize(int fit_index, bool padded)
{
  int size = sizeof(unsigned int) * _fd[fit_index]->GetNumParamsToFit();  
  return (!padded)?(size):(padTo128Bytes(size));
}

int WorkSet::getParamMatrixSize(int fit_index, bool padded)
{
  int size = ((_fd[fit_index]->GetNumParamsToFit()*_fd[fit_index]->GetNumParamsToFit()+ 1)/2)*sizeof(float);
  return size * ((!padded)?(getNumBeads()):(getPaddedN()));
}

int WorkSet::getParamRHSSize(int fit_index, bool padded)
{
  int size = _fd[fit_index]->GetNumParamsToFit() *sizeof(float);
  return size * ((!padded)?(getNumBeads()):(getPaddedN()));
}




int WorkSet::getPartialDerivStepsMaxSize(bool padded)
{
  int size = sizeof(CpuStep_t)*getMaxSteps();
  return (!padded)?(size):(padTo128Bytes(size));
}

int WorkSet::getJTJMatrixMapMaxSize(bool padded)
{
  int size = sizeof(unsigned int) * getMaxParams()*getMaxParams();  
  return (!padded)?(size):(padTo128Bytes(size));
}

int WorkSet::getBeadParamIdxMapMaxSize(bool padded) 
{
  int size = sizeof(unsigned int) * getMaxParams();
  return (!padded)?(size):(padTo128Bytes(size));
}

int WorkSet::getParamMatrixMaxSize(bool padded)
{
  int size = ((getMaxParams()*(getMaxParams() + 1))/2)*sizeof(float);
  return size * ((!padded)?(getNumBeads()):(getPaddedN()));
}

int WorkSet::getParamRHSMaxSize(bool padded)
{
  int size = getMaxParams() *sizeof(float);
  return size * ((!padded)?(getNumBeads()):(getPaddedN()));
}

int WorkSet::getFrameNumberSize(bool padded)
{
	int size = sizeof(float)*getNumFrames();
	return (!padded)?(size):(padTo128Bytes(size));
}

//////////////////////////////////////////////////////////////////////////////////////
///

int WorkSet::getPaddedN(){
  return ((getNumBeads()+32-1)/32)*32;
}


int WorkSet::padTo128Bytes(int size){
  return ((size+128-1)/128)*128;
}


bool WorkSet::ValidJob()
{
  
  if (_info->bkgObj->region_data->fitters_applied == -1 || _info == NULL) {
    return false;
  }
  return isSet();

}

void WorkSet::KeyNormalize()
{
  _info->bkgObj->region_data->my_beads.my_mean_copy_count = _info->bkgObj->region_data->my_beads.KeyNormalizeReads(true); 
}

void WorkSet::PerformePCA()
{
  _info->bkgObj->CPU_DarkMatterPCA();
}

void WorkSet::setJobToPostFitStep()
{
    _info->type = POST_FIT_STEPS;
    _info->bkgObj->region_data->fitters_applied=TIME_TO_DO_PREWELL;
}

void WorkSet::setJobToRemainRegionFit()
{
  _info->type = INITIAL_FLOW_BLOCK_REMAIN_REGIONAL_FIT;
  _info->bkgObj->region_data->fitters_applied=TIME_TO_DO_REMAIN_MULTI_FLOW_FIT_STEPS;
}

void WorkSet::putJobToCPU(WorkerInfoQueueItem item)
{
  _info->pq->GetCpuQueue()->PutItem(item);
}

void WorkSet::putJobToGPU(WorkerInfoQueueItem item)
{
  _info->pq->GetGpuQueue()->PutItem(item);
}

void WorkSet::printJobSummary()
{

  if( ValidJob() )
  {
    cout << " | Job Summary:" << endl
    << " | max beads: " << GpuMultiFlowFitControl::GetMaxBeads() << " max frames: " << GpuMultiFlowFitControl::GetMaxFrames() << endl
    << " | live beads: " << getNumBeads() <<" padded: "<< getPaddedN()  << endl
    << " | num frames: " << getNumFrames() << endl
    << " | flow num:   " << getAbsoluteFlowNum() << endl
   << " +------------------------------" << endl
    ; 
  }
  else{
   cout << "No Valid Job Set" << endl;
  }
  
}

int WorkSet::getXtalkNeiIdxMapSize(bool padded)
{
  int size = sizeof(int) * MAX_XTALK_NEIGHBOURS;  
  return size*( (!padded)?(getNumBeads()):(getPaddedN()) ); 
}

int WorkSet::getNumXtalkNeighbours() {
  return _info->bkgObj->getXtalkExecute().xtalk_spec_p->nei_affected;  
}

const int* WorkSet::getNeiIdxMapForXtalk() {
  return _info->bkgObj->getXtalkExecute().GetNeighborIndexMap();
}

int* WorkSet::getXtalkNeiXCoords() {
  return &_info->bkgObj->getXtalkExecute().xtalk_spec_p->cx[0];  
}

int* WorkSet::getXtalkNeiYCoords() {
  return &_info->bkgObj->getXtalkExecute().xtalk_spec_p->cy[0];  
}

float* WorkSet::getXtalkNeiMultiplier() {
  return &_info->bkgObj->getXtalkExecute().xtalk_spec_p->multiplier[0];  
}

float* WorkSet::getXtalkNeiTauTop() {
  return &_info->bkgObj->getXtalkExecute().xtalk_spec_p->tau_top[0];  
}

float* WorkSet::getXtalkNeiTauFluid() {
  return &_info->bkgObj->getXtalkExecute().xtalk_spec_p->tau_fluid[0];  
}

void WorkSet::calculateCPUXtalkForBead(int ibd, float* buf) {
  _info->bkgObj->getXtalkExecute().ExecuteXtalkFlux(ibd, buf); 
}

bool WorkSet::performCrossTalkCorrection() {
  return _info->bkgObj->getXtalkExecute().xtalk_spec_p->do_xtalk_correction;
}

bool WorkSet::performExpTailFitting() {
  return _info->bkgObj->getGlobalDefaultsForBkgModel().signal_process_control.exp_tail_fit;
}

bool WorkSet::performCalcPCADarkMatter() {
  return _info->bkgObj->getGlobalDefaultsForBkgModel().signal_process_control.pca_dark_matter;
}

bool WorkSet::useDarkMatterPCA() {
  return (  performCalcPCADarkMatter() && _info->bkgObj->region_data->my_regions.missing_mass.mytype == PCAVector)?(true):(false) ;
}


